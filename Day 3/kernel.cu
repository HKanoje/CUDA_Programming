#include "hip/hip_runtime.h"
﻿#include <iostream>
#include <hip/hip_runtime.h>

// CUDA Kernel for Matrix-Vector Multiplication
__global__ void VectorMatrixMul(const float* A, const float* B, float* C, int N) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N) {
        float sum = 0.0f;
        for (int j = 0; j < N; j++) {  // Corrected loop condition
            sum += A[j * N + i] * B[j]; // Multiplying matrix column with vector
        }
        C[i] = sum;
    }
}

int main() {
    // Define Matrix/Vector Size
    const int N = 3;
    float* A, * B, * C;

    // Allocate Host Memory
    A = (float*)malloc(N * N * sizeof(float));
    B = (float*)malloc(N * sizeof(float));
    C = (float*)malloc(N * sizeof(float));

    // Initialize Matrix A (All 1s) and Vector B (All 2s)
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            A[i * N + j] = 1.0f;
        }
        B[i] = 2.0f;
        C[i] = 0.0f;
    }

    // Allocate Device Memory
    float* d_A, * d_B, * d_C;
    hipMalloc(&d_A, N * N * sizeof(float));
    hipMalloc(&d_B, N * sizeof(float));
    hipMalloc(&d_C, N * sizeof(float));

    // Copy Data from Host to Device
    hipMemcpy(d_A, A, N * N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, N * sizeof(float), hipMemcpyHostToDevice);

    // Configure Kernel Launch
    int blocksize = 256;
    int gridsize = (N + blocksize - 1) / blocksize;

    // Measure Execution Time
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);  // Start Timing

    // Launch Kernel
    VectorMatrixMul << <gridsize, blocksize >> > (d_A, d_B, d_C, N);

    hipEventRecord(stop);   // Stop Timing
    hipEventSynchronize(stop);

    // Copy Result Back to Host
    hipMemcpy(C, d_C, N * sizeof(float), hipMemcpyDeviceToHost);

    // Calculate Time Taken
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);



    // Print GPU Details
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    printf("GPU Used: %s\n", prop.name);
    // Print Execution Time
    printf("Kernel Execution Time: %.5f ms\n", milliseconds);
    printf("\n");

    // Print Matrix A
    printf("Matrix A:\n");
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            printf("%.2f ", A[i * N + j]);
        }
        printf("\n");
    }

    // Print Vector B
    printf("\nMatrix B:\n");
    for (int i = 0; i < N; i++) {
        printf("%.2f ", B[i]);
    }
    printf("\n");

    // Print Result Vector C
    printf("\nResult Vector C (A * B):\n");
    for (int i = 0; i < N; i++) {
        printf("%.2f ", C[i]);
    }
    printf("\n");



    // Free Allocated Memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    free(A);
    free(B);
    free(C);

    return 0;
}
