#include "hip/hip_runtime.h"
﻿#include <iostream>
#include <hip/hip_runtime.h>

__global__ void vectorAdd(const float* A, const float* B, float* C, int N) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N) {
        C[i] = A[i] + B[i];
    }
}

int main() {
    const int N = 10;
    float A[N], B[N], C[N];

    // Initialize arrays
    for (int i = 0; i < N; i++) {
        A[i] = static_cast<float>(i);
        B[i] = static_cast<float>(i * 2);
    }

    float* d_a, * d_b, * d_c;
    hipMalloc(&d_a, N * sizeof(float));
    hipMalloc(&d_b, N * sizeof(float));
    hipMalloc(&d_c, N * sizeof(float));

    hipMemcpy(d_a, A, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, B, N * sizeof(float), hipMemcpyHostToDevice);

    int blocksize = 256;
    int gridsize = (N + blocksize - 1) / blocksize; // Ensure proper grid size

    // Get GPU Name
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0); // Assuming the first GPU (device 0) is used
    std::cout << "Running on GPU: " << prop.name << std::endl;

    // Start timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    // Launch kernel
    vectorAdd << <gridsize, blocksize >> > (d_a, d_b, d_c, N);

    // Stop timing
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    hipMemcpy(C, d_c, N * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    // Print result
    for (int i = 0; i < N; i++) {
        std::cout << "C[" << i << "] = " << C[i] << std::endl;
    }

    // Print execution time
    std::cout << "Kernel execution time: " << milliseconds << " ms" << std::endl;

    return 0;
}
